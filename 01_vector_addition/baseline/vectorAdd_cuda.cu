#include "hip/hip_runtime.h"
// This program computes the sum of two vectors of length N
// By: Nick from CoffeeBeforeArch

#include <cassert>
#include <vector>
#include <algorithm>

namespace cuda
{
namespace __impl
{
// CUDA kernel for vector addition
// __global__ means this is called from the CPU, and runs on the GPU
__global__ void vectorAdd(const int *__restrict a, const int *__restrict b,
                          int *__restrict c, int N) {
  // Calculate global thread ID
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

  // Boundary check
  if (tid < N) c[tid] = a[tid] + b[tid];
}
} // __impl

std::vector<int> vectorAdd(const std::vector<int> &input_1,
                           const std::vector<int> &input_2){
  //validate input is coorectly sized.
  //All input vectors must be of same size
  assert(input_1.size() == input_2.size());
  // Allocate memory on the device
  size_t input_size = input_1.size();
  std::vector<int> output(input_size, 0);
  int *d_input_1, *d_input_2, *d_output;
  size_t bytes = sizeof(int) * input_size;
  hipMalloc(&d_input_1, bytes);
  hipMalloc(&d_input_2, bytes);
  hipMalloc(&d_output, bytes);

  // Copy data from the host to the device (CPU -> GPU)
  hipMemcpy(d_input_1, input_1.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_input_2, input_2.data(), bytes, hipMemcpyHostToDevice);

  // Threads per CTA (1024)
  int NUM_THREADS = 1 << 10;

  // CTAs per Grid
  // We need to launch at LEAST as many threads as we have elements
  // This equation pads an extra CTA to the grid if N cannot evenly be divided
  // by NUM_THREADS (e.g. N = 1025, NUM_THREADS = 1024)
  int NUM_BLOCKS = (input_size + NUM_THREADS - 1) / NUM_THREADS;

  // Launch the kernel on the GPU
  // Kernel calls are asynchronous (the CPU program continues execution after
  // call, but no necessarily before the kernel finishes)
  __impl::vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(d_input_1, d_input_2, d_output, input_size);

  // Copy sum vector from device to host
  // hipMemcpy is a synchronous operation, and waits for the prior kernel
  // launch to complete (both go to the default stream in this case).
  // Therefore, this hipMemcpy acts as both a memcpy and synchronization
  // barrier.
  hipMemcpy(output.data(), d_output, bytes, hipMemcpyDeviceToHost);

  // Free memory on device
  hipFree(d_input_1);
  hipFree(d_input_2);
  hipFree(d_output);
  return output;
}

std::vector<int> vectorAdd(const std::vector<std::vector<int>> &inputs){
  size_t inputs_size = inputs.size();
  assert(inputs_size > 0);
  const size_t input_size = inputs[0].size();
  for(int i = 0; i < inputs_size; i++){
      assert(inputs[i].size() == input_size);
  }
  std::vector<int> output = inputs[0];
  for(auto itr = inputs.cbegin() + 1;itr != inputs.cend(); itr++){
      output = vectorAdd(output, *itr);
  }
  return output;
}

std::vector<int> vectorAdd_O1(const std::vector<std::vector<int>> &inputs)
{
  size_t inputs_size = inputs.size();
  assert(inputs_size > 0);
  const size_t input_size = inputs[0].size();
  for(int i = 0; i < inputs_size; i++){
      assert(inputs[i].size() == input_size);
  }

  std::vector<int> output(input_size, 0);
  int *d_input_1, *d_input_2, *d_output;
  size_t bytes = sizeof(int) * input_size;
  int NUM_THREADS = 1 << 10;
  int NUM_BLOCKS = (input_size + NUM_THREADS - 1) / NUM_THREADS;

  hipMalloc(&d_input_1, bytes);
  hipMalloc(&d_input_2, bytes);
  hipMalloc(&d_output, bytes);

  hipMemcpy(d_input_1, inputs[0].data(), bytes, hipMemcpyHostToDevice);
  for(auto itr = inputs.cbegin() + 1; itr != inputs.cend(); itr++) {
    hipMemcpy(d_input_2, (*itr).data(), bytes, hipMemcpyHostToDevice);
    __impl::vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(d_input_1, d_input_2, d_output, input_size);
    d_input_1 = d_output;
  }

  hipMemcpy(output.data(), d_input_1, bytes, hipMemcpyDeviceToHost);

  // Free memory on device
  hipFree(d_input_1);
  hipFree(d_input_2);
  hipFree(d_output);
  return output;
}

std::vector<int> vectorAdd_O2(const std::vector<std::vector<int>> &inputs){
  size_t input_count = inputs.size();
  assert(input_count > 0);
  const size_t input_size = inputs[0].size();
  size_t bytes = sizeof(int) * input_size;
  for(int i = 0; i < input_count; i++) {
      assert(inputs[i].size() == input_size);
  }

  std::vector<int> output(input_size, 0);
  int **pd_input;
  hipMalloc(&pd_input, input_count * sizeof(int*));
  for(int i = 0; i < input_count; i++){
    hipMalloc(&pd_input[i], bytes);
    hipMemcpy(pd_input[i], inputs[i].data(), bytes, hipMemcpyHostToDevice);
  }

  int NUM_THREADS = 1 << 10;
  int NUM_BLOCKS = (input_size + NUM_THREADS - 1) / NUM_THREADS;
  int BS = 1;
  while(true)
  {
    int a = 0, b = a + BS, step = 2*BS;
    bool computed = false;
    while(a < input_count && b < input_count)
    {
      __impl::vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(pd_input[a], pd_input[b], pd_input[a], input_size);
      a += step;
      b += step;
    }
    if(!computed){
      break;
    }
    BS << 1;
  }
  hipMemcpy(output.data(), d_input[0], bytes, hipMemcpyDeviceToHost);
  for(int i = 0; i < input_count; i++){
    hipFree(pd_input[i]);
  }
  hipFree(pd_input);
  return output;
}
} //cuda
